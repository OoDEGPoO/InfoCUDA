#include "hip/hip_runtime.h"
//CUDA
#include "hip/hip_runtime.h"
#include ""
#include "D:\Desarrollo\CUDA\CUDA-By-Example-master\common\book.h"

//Salida
#include <fstream>
#include <iostream>
using namespace std;
#include <stdio.h>

int main(void) {

	//Diego-Edgar Gracia Pe�a
	//Fichero de salida
	ofstream salida;
	salida.open("salida.txt");

	//dispositivo CUDA
	hipDeviceProp_t prop;
	int count, driverVersion = 0, runtimeVersion = 0;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count;i++) {
		hipGetDeviceProperties(&prop, i);
		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		printf(" Nombre: %s \n", prop.name);
		printf(" --Driver de CUDA: %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
		printf(" --Version Runtime: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
		printf(" --major y minior : %d,%d  \n", prop.major, prop.minor);
		printf(" --pciBusID: %d \n", prop.pciBusID);
		printf(" --pciDeviceID : %d \n", prop.pciDeviceID);
		printf(" --tccDriver : %d \n", prop.tccDriver);
		printf(" Frecuencia de reloj: %d \n", prop.clockRate);
		printf(" Numero de Multiprocesadores: %d \n", prop.multiProcessorCount);
		printf(" Capacidad Bus de Memoria: %d bits \n", prop.memoryBusWidth);
		printf(" Frecuencia de memoria: %d \n", prop.memoryClockRate);
		printf(" dimension warp: %d  \n", prop.warpSize);
		printf(" registros por bloque %d  \n", prop.regsPerBlock);
		printf(" memoria global total: %u \n", prop.totalGlobalMem);
		printf(" memoria compartida por bloque: %u \n", prop.sharedMemPerBlock);
		printf(" pitch: %u \n", prop.memPitch);//DEGP
		printf(" max hilos por bloque %d \n", prop.maxThreadsPerBlock);
		printf(" max dimension de bloque: %d \n", prop.maxThreadsDim);
		printf(" max dimension de grid: %d \n", prop.maxGridSize);
		printf(" total ConstMem: %u \n", prop.totalConstMem);
		printf(" alineamiento de textura: %u \n", prop.textureAlignment);
		printf(" deviceoverlap : %d \n", prop.deviceOverlap);
		printf(" multiProcessorCount : %d \n", prop.multiProcessorCount);
		printf(" kernelExecTimeoutEnabled : %d \n", prop.kernelExecTimeoutEnabled);
		printf(" integrated : %d \n", prop.integrated);
		printf(" canMapHostMemory : %d \n", prop.canMapHostMemory);
		printf(" computeMode : %d \n", prop.computeMode);
		printf(" concurrentKernels : %d \n", prop.concurrentKernels);
		printf(" ECCEnabled: %d \n", prop.ECCEnabled);

		salida << i << "\n";
		salida << " Nombre: " << prop.name << " \n";
		salida << " --Driver de CUDA: " << driverVersion / 1000 << "." << (driverVersion % 100) / 10 << "\n";
		salida << " --Version Runtime: " << runtimeVersion / 1000 << "." << (runtimeVersion % 100) / 10 << "\n";
		salida << " --major y minior : " << prop.major << "," << prop.minor << "\n";
		salida << " --pciBusID: " << prop.pciBusID << "\n";
		salida << " --pciDeviceID : " << prop.pciDeviceID << "\n";
		salida << " --tccDriver : " << prop.tccDriver << "\n";
		salida << " Frecuencia de reloj: " << prop.clockRate << "\n";
		salida << " Numero de Multiprocesadores: " << prop.multiProcessorCount << " \n";
		salida << " Capacidad Bus de Memoria: " << prop.memoryBusWidth << " bits\n";
		salida << " Frecuencia de memoria: " << prop.memoryClockRate << "\n";
		salida << " dimension de warp: " << prop.warpSize << "\n";
		salida << " registros por bloque " << prop.regsPerBlock << "\n";
		salida << " memoria global total: " << prop.totalGlobalMem << "\n";
		salida << " memoria compartida por bloque: " << prop.sharedMemPerBlock << "\n";
		salida << " pitch: " << prop.memPitch << "\n";
		salida << " max hilos por bloque " << prop.maxThreadsPerBlock << "\n";
		salida << " max dimension de bloque: " << prop.maxThreadsDim << "\n";//D
		salida << " max dimension de grid: " << prop.maxGridSize << "\n";//E
		salida << " total ConstMem: " << prop.totalConstMem << "\n";//G
		salida << " alineamiento de textura: " << prop.textureAlignment << "\n";//P
		salida << " deviceoverlap : " << prop.deviceOverlap << "\n";
		salida << " multiProcessorCount : " << prop.multiProcessorCount << "\n";
		salida << " kernelExecTimeoutEnabled : " << prop.kernelExecTimeoutEnabled << "\n";
		salida << " integrated : " << prop.integrated << "\n";
		salida << " canMapHostMemory : " << prop.canMapHostMemory << "\n";
		salida << " computeMode : " << prop.computeMode << "\n";
		salida << " concurrentKernels : " << prop.concurrentKernels << "\n";
		salida << " ECCEnabled: " << prop.ECCEnabled << "\n";
	}

	salida.close();
	//char *s;
	//scanf("%c", s);

	return 0;
}